#include <chrono>
#include <iomanip>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call_)                                                                                                                                    \
        {                                                                                                                                                          \
                hipError_t status = call_;                                                                                                                  \
                if(status != hipSuccess) {                                                                                                                            \
                        fprintf(stderr, "CUDA Error in call %s on line %d: %s\n", #call_, __LINE__, hipGetErrorString(status));                                              \
                        abort();                                                                                                                                           \
                }                                                                                                                                                      \
        }

struct Metric {
    double min = std::numeric_limits<double>::max();
    double max = 0;
    double avg = 0;

    void update(double value) {
        min = std::min(min, value);
        max = std::max(max, value);
        avg += value;
    }
};

struct CombinedMetric {
    Metric individual;
    Metric concurrent;
};

struct Device {
    int id;
    hipStream_t stream;
    int *d_data;
    int *h_data;
    int *d_staging;
    int *h_staging;
    CombinedMetric d2h_bandwidth;
    CombinedMetric d2h_bw_strided;
    CombinedMetric d2h_bw_d_strided;
    CombinedMetric d2h_bw_h_strided;
    CombinedMetric d2h_bw_strided_kernel;
    CombinedMetric h2d_bandwidth;
    CombinedMetric h2d_bw_strided;
    CombinedMetric h2d_bw_d_strided;
    CombinedMetric h2d_bw_h_strided;
};

// kernel to gather strided date into contiguous memory
__global__ void linearize(int *src, int *dst, size_t stride, size_t size) {
    size_t idx = (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < size) {
        dst[idx] = src[idx * stride];
    }
}

void host_linearize(int *src, int *dst, size_t stride, size_t size) {
    for (size_t i = 0; i < size; i++) {
        dst[i] = src[i * stride];
    }
}

// kernel to scatter contiguous data into strided memory
__global__ void delinearize(int *src, int *dst, size_t stride, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx * stride] = src[idx];
    }
}

void host_delinearize(int *src, int *dst, size_t stride, size_t size) {
    for (size_t i = 0; i < size; i++) {
        dst[i * stride] = src[i];
    }
}

int main(int argc, char **argv) {
    int repeats = 10;
    const int warmups = 3;
    if (argc > 1) {
        repeats = atol(argv[1]);
    }
    size_t memStride = 4096;
    if (argc > 2) {
        memStride = atol(argv[2]);
    }
    size_t transferSize = 1024 * 1024 * sizeof(int);
    if (argc > 3) {
        transferSize = atol(argv[3]);
    }
    std::cout << "Repeats: " << repeats << ", Stride: " << memStride << ", Transfer Size: " << transferSize << std::endl;

    using namespace std::chrono_literals;

    // init CUDA devices
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA Device Count: " << deviceCount << std::endl;

    // print device info
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device " << i << ": " << prop.name << std::endl;
    }

    const size_t bytesTotal = memStride * transferSize;
    std::cout << "Allocating " << (bytesTotal) / (1024 * 1024) << " MB of memory per device as main buffer, and " << (transferSize) / (1024 * 1024) << " MB as staging buffer" << std::endl;

    // initialize device structs
    std::vector<Device> devices(deviceCount);
    for (int i = 0; i < deviceCount; i++) {
        devices[i].id = i;
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipStreamCreate(&devices[i].stream));
        CUDA_CHECK(hipMalloc(&devices[i].d_data, bytesTotal));
        CUDA_CHECK(hipHostMalloc(&devices[i].h_data, bytesTotal, hipHostMallocDefault));
        CUDA_CHECK(hipMalloc(&devices[i].d_staging, transferSize));
        CUDA_CHECK(hipHostMalloc(&devices[i].h_staging, transferSize, hipHostMallocDefault));
    }

    const auto measureIndividual = [&](auto operation, auto metric) {
        for (int r = 0; r < repeats+warmups; ++r) {
            for (int i = 0; i < deviceCount; i++) {
                CUDA_CHECK(hipStreamSynchronize(devices[i].stream));
                const auto start = std::chrono::high_resolution_clock::now();
                operation(i);
                CUDA_CHECK(hipStreamSynchronize(devices[i].stream));
                const auto end = std::chrono::high_resolution_clock::now();
                const auto elapsed = end - start;
                if(r>=warmups) {
                    double current_bw = (double)transferSize / (1024 * 1024) / (elapsed / 1.0s);
                    metric(i).individual.update(current_bw);
                }
            }
        }
    };
    
    const auto measureConcurrent = [&](auto operation, auto metric) {
        for (int r = 0; r < repeats+warmups; ++r) {
            for (int i = 0; i < deviceCount; i++) {
                CUDA_CHECK(hipStreamSynchronize(devices[i].stream));
            }
            const auto start = std::chrono::high_resolution_clock::now();
            for (int i = 0; i < deviceCount; i++) {
                operation(i);
            }
            for (int i = 0; i < deviceCount; i++) {
                CUDA_CHECK(hipStreamSynchronize(devices[i].stream));
            }
            const auto end = std::chrono::high_resolution_clock::now();
            const auto elapsed = end - start;
            if(r>=warmups) {
                double current_bw = (double)transferSize / (1024 * 1024) / (elapsed / 1.0s);
                for (int i = 0; i < deviceCount; i++) {
                    metric(i).concurrent.update(current_bw);
                }
            }
        }
    };

    auto measureBoth = [&](auto operation, auto metric) {
        measureIndividual(operation, metric);
        measureConcurrent(operation, metric);
    };

    // measure device to host bandwidth with contiguous access
    measureBoth([&](int i) {
        CUDA_CHECK(hipMemcpyAsync(devices[i].h_data, devices[i].d_data, transferSize, hipMemcpyDeviceToHost, devices[i].stream));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].d2h_bandwidth;
    });

    // measure device to host bandwidth with strided access on both
    measureBoth([&](int i) {
        CUDA_CHECK(hipMemcpy2DAsync(devices[i].h_data, memStride, devices[i].d_data, memStride, 
            sizeof(int), transferSize/sizeof(int), hipMemcpyDeviceToHost, devices[i].stream));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].d2h_bw_strided;
    });

    // measure device to host bandwidth with strided access on device
    measureBoth([&](int i) {
        CUDA_CHECK(hipMemcpy2DAsync(devices[i].h_data, sizeof(int), devices[i].d_data, memStride, 
            sizeof(int), transferSize/sizeof(int), hipMemcpyDeviceToHost, devices[i].stream));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].d2h_bw_d_strided;
    });

    // measure device to host bandwidth with strided access on host
    measureBoth([&](int i) {
        CUDA_CHECK(hipMemcpy2DAsync(devices[i].h_data, memStride, devices[i].d_data, sizeof(int), 
            sizeof(int), transferSize/sizeof(int), hipMemcpyDeviceToHost, devices[i].stream));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].d2h_bw_h_strided;
    });

    // measure device to host bandwidth with strided access using kernel linearize / delinearize
    measureBoth([&](int i) {
        linearize<<<(transferSize/sizeof(int) + 255) / 256, 256, 0, devices[i].stream>>>(devices[i].d_data, devices[i].d_staging, memStride/sizeof(int), transferSize/sizeof(int));
        CUDA_CHECK(hipMemcpyAsync(devices[i].h_staging, devices[i].d_staging, transferSize, hipMemcpyDeviceToHost, devices[i].stream));
        hipStreamSynchronize(devices[i].stream);
        host_delinearize(devices[i].h_staging, devices[i].h_data, memStride/sizeof(int), transferSize/sizeof(int));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].d2h_bw_strided_kernel;
    });

    // measure host to device bandwidth with contiguous access
    measureBoth([&](int i) {
        CUDA_CHECK(hipMemcpyAsync(devices[i].d_data, devices[i].h_data, transferSize, hipMemcpyHostToDevice, devices[i].stream));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].h2d_bandwidth;
    });

    // measure host to device bandwidth with strided access on both
    measureBoth([&](int i) {
        CUDA_CHECK(hipMemcpy2DAsync(devices[i].d_data, memStride, devices[i].h_data, memStride, 
            sizeof(int), transferSize/sizeof(int), hipMemcpyHostToDevice, devices[i].stream));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].h2d_bw_strided;
    });

    // measure host to device bandwidth with strided access on device
    measureBoth([&](int i) {
        CUDA_CHECK(hipMemcpy2DAsync(devices[i].d_data, memStride, devices[i].h_data, sizeof(int), 
            sizeof(int), transferSize/sizeof(int), hipMemcpyHostToDevice, devices[i].stream));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].h2d_bw_d_strided;
    });

    // measure host to device bandwidth with strided access on host
    measureBoth([&](int i) {
        CUDA_CHECK(hipMemcpy2DAsync(devices[i].d_data, sizeof(int), devices[i].h_data, memStride, 
            sizeof(int), transferSize/sizeof(int), hipMemcpyHostToDevice, devices[i].stream));
    }, [&](int i) -> CombinedMetric& {
        return devices[i].h2d_bw_h_strided;
    });

    std::cout << std::fixed << std::setprecision(1);
    const auto printMetric = [&](const char* name, auto metric) {
        for(int i=0; i<deviceCount; i++) {
            std::cout << std::setw(24) << name << " -   Device " << i << " " << ": " << std::setw(9) << metric(i).individual.min << " MB/s (min), " << std::setw(9) << metric(i).individual.max << " MB/s (max), " << std::setw(9) << metric(i).individual.avg / repeats << " MB/s (avg)" << std::endl;
        }
        std::cout << std::setw(24) << name << " - Concurrent " << ": " << std::setw(9) << metric(0).concurrent.min << " MB/s (min), " << std::setw(9) << metric(0).concurrent.max << " MB/s (max), " << std::setw(9) << metric(0).concurrent.avg / repeats << " MB/s (avg)" << std::endl;
    };

    printMetric("D2H Contiguous", [&](int i) -> const CombinedMetric& {
        return devices[i].d2h_bandwidth;
    });
    printMetric("D2H Strided Both", [&](int i) -> const CombinedMetric& {
        return devices[i].d2h_bw_strided;
    });
    printMetric("D2H Strided Device", [&](int i) -> const CombinedMetric& {
        return devices[i].d2h_bw_d_strided;
    });
    printMetric("D2H Strided Host", [&](int i) -> const CombinedMetric& {
        return devices[i].d2h_bw_h_strided;
    });
    printMetric("D2H Strided w/ Kernel", [&](int i) -> const CombinedMetric& {
        return devices[i].d2h_bw_strided_kernel;
    });
    printMetric("H2D Contiguous", [&](int i) -> const CombinedMetric& {
        return devices[i].h2d_bandwidth;
    });
    printMetric("H2D Strided Both", [&](int i) -> const CombinedMetric& {
        return devices[i].h2d_bw_strided;
    });
    printMetric("H2D Strided Device", [&](int i) -> const CombinedMetric& {
        return devices[i].h2d_bw_d_strided;
    });
    printMetric("H2D Strided Host", [&](int i) -> const CombinedMetric& {
        return devices[i].h2d_bw_h_strided;
    });

    // free device structs
    for (int i = 0; i < deviceCount; i++) {
        hipSetDevice(i);
        hipFree(devices[i].d_data);
        hipHostFree(devices[i].h_data);
        hipStreamDestroy(devices[i].stream);
    }
}
